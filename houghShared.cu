#include "hip/hip_runtime.h"
/*============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <vector>
#include <jpeglib.h>
#include <string>
#include "pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float* d_Cos, float* d_Sin)
{
  //TODO calcular: int gloID = ?
  int gloID = ( blockIdx.x ) * blockDim.x +
                threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}

//*****************************************************************


//SHARED MEMORY
// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTranshared (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float* d_Cos, float* d_Sin)
{
  // Calculate global ID
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  // Use shared memory here for acc variable
  __shared__ int localAcc[degreeBins * rBins];
  // Initialize
  for (int i = threadIdx.x; i < degreeBins * rBins; i += blockDim.x)
    localAcc[i] = 0;

  // Synchronize threads
  __syncthreads();

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;

          // We perform atomicAdd on shared memory
          atomicAdd (&localAcc[rIdx * degreeBins + tIdx], 1);
        }
    }
  
  // Synchronize threads
  __syncthreads();

  // Adding from shared to global memory
  for (int i = threadIdx.x; i < degreeBins * rBins; i += blockDim.x)
    atomicAdd (&acc[i], localAcc[i]);
}

//***********************************************************
int main (int argc, char **argv)
{
  int i;
  std::string arg = argv[2];
  std::size_t pos;
  int threshold = std::stoi(arg,&pos);

  PGMImage* inImg = new PGMImage(argv[1], 2);

  int *cpuht;
  int w = inImg->getXDim();
  int h = inImg->getYDim();

  float* d_Cos;
  float* d_Sin;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg->getPixels(), w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // TODO eventualmente volver memoria global
  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg->getPixels(); // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);
  hipEventRecord(start);
  GPU_HoughTranshared <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);
  hipEventRecord(stop);
  
  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i]) {
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }
  }
  printf("Done!\n");
  std::vector<std::pair<int, int>> lines;
  for (i = 0; i < degreeBins * rBins; i++){
    if (h_hough[i] > threshold) {
      // pair order: r, th
      int my_r = i / degreeBins;
      int my_th = i % degreeBins;
      std::pair<int, int> line = {my_r, my_th};
      lines.push_back(line);
    }
  }
  inImg->write("SharedOutput.jpeg", lines, radInc, rBins);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Milliseconds: %.3f ms\n" ,milliseconds);
  printf("Seconds: %d.%.3d s\n", (int)milliseconds/1000, (int)milliseconds%1000);

  hipFree((void *) d_Cos);
  hipFree((void *) d_Sin);
  //add
  hipFree((void *) d_in);
	hipFree((void *) d_hough);
  delete[] pcCos;
  delete[] pcSin;
  delete inImg;
  hipDeviceReset();

  return 0;
}